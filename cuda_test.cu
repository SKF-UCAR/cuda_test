#include <stdio.h>
#include <hip/hip_runtime_api.h>

int main(int argc, char** argv )
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));
    printf("Device Count:", deviceCount);


    return 0;
}