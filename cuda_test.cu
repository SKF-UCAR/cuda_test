#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(error_id) << std::endl;
        return EXIT_FAILURE;
    }

    if (deviceCount == 0) {
        std::cout << "No CUDA-capable devices detected." << std::endl;
        return EXIT_SUCCESS;
    }

    std::cout << "Number of CUDA-capable devices: " << deviceCount << std::endl;

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        std::cout << "\nDevice " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Maximum Threads per Block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Maximum Threads per Multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "  Maximum Grid Size: [" 
                  << deviceProp.maxGridSize[0] << ", " 
                  << deviceProp.maxGridSize[1] << ", " 
                  << deviceProp.maxGridSize[2] << "]" << std::endl;
        std::cout << "  Maximum Threads Dimensions: [" 
                  << deviceProp.maxThreadsDim[0] << ", " 
                  << deviceProp.maxThreadsDim[1] << ", " 
                  << deviceProp.maxThreadsDim[2] << "]" << std::endl;
    }

    return EXIT_SUCCESS;
}
